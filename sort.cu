#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <time.h>


#define THREADS_PER_BLOCK 1024

// function to sort blocks of size <THREADS_PER_BLOCK> which will then be parallel merged with another call from the host
// this one is optimized with shared memory

__global__ void shared_bitonic_block_sort(int* arr, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // copy into shared memory for reduced global memory accesses
    extern __shared__ int shared_arr[];
    if (index < n) {
        shared_arr[tid] = arr[index];
    }
    __syncthreads();
    // Bitonic sort in shared memory
    for (int size = 2; size <= blockDim.x; size = size << 1) {
        for (int stride = size >> 1; stride >= 1; stride = stride >> 1) {
            int partner_index = tid ^ stride; 
            if (partner_index > tid) {  
                bool descending = ((size & tid) != 0);
                // Array is supposed to be increasing at this point
                if (!descending && shared_arr[tid] > shared_arr[partner_index]) {
                    int temp = shared_arr[tid];
                    shared_arr[tid] = shared_arr[partner_index];
                    shared_arr[partner_index] = temp;
                }
                // Array is supposed to be decreasing at this point
                else if (descending && shared_arr[tid] < shared_arr[partner_index]) {
                    int temp = shared_arr[tid];
                    shared_arr[tid] = shared_arr[partner_index];
                    shared_arr[partner_index] = temp;
                }
            }
            __syncthreads();
        }
    }
    if (index < n) {
        arr[index] = shared_arr[tid];
    }
}

// a compare and swap method unique to bitonic sort that considers whether the array should be increasing or decreasing at this point
__device__ void bitonic_swap(int* arr, int cur_index, int partner_index, bool descending) {
    // Array is supposed to be increasing at this point
    if (!descending && arr[cur_index] > arr[partner_index]) {
        int temp = arr[cur_index];
        arr[cur_index] = arr[partner_index];
        arr[partner_index] = temp;
    }
    // Array is supposed to be decreasing at this point
    else if (descending && arr[cur_index] < arr[partner_index]) {
        int temp = arr[cur_index];
        arr[cur_index] = arr[partner_index];
        arr[partner_index] = temp;
    }
}

// function to sort blocks of size <THREADS_PER_BLOCK> which will then be parallel merged with another call from the host

__global__ void bitonic_block_sort(int* arr, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Bitonic sort in global memory
    for (int size = 2; size <= blockDim.x; size = size << 1) {
        for (int stride = size >> 1; stride >= 1; stride = stride >> 1) {
            int partner_index = index ^ stride; 
            if (partner_index > index) {  
                bool descending = ((size & threadIdx.x) != 0);
                bitonic_swap(arr, index, partner_index, descending);
            }
            __syncthreads();
        }
    }
}

// cuda kernel to get the input range that each thread needs to look at
// based on binary search
// Adapted from: Programming Massively Parallel Processors A Hands-on Approach

__device__ int co_rank(int k, int* A, int m, int* B, int n){
    int i = k < m ? k : m; // i = min(k, m)
    int j = k - i;
    int i_low = 0 > (k - n) ? 0 : k - n;
    int j_low = 0 > (k - m) ? 0 : k - m;
    int delta;
    bool active = true;

    while (active) {
        if (i > 0 && j < n && A[i - 1] > B[j]) {
            delta = ((i - i_low + 1) >> 1);
            j_low = j;
            j = j + delta;
            i = i - delta;
        } else if (j > 0 && i < m && B[j - 1] >= A[i]) {
            delta = ((j - j_low + 1) >> 1); 
            i_low = i;
            i = i + delta;
            j = j - delta;
        } else {
            active = false;
        }
    }

    return i;
}

// cuda kernel for one thread to sequentially merge A with B and place in C
// assumes A and B chunks are already sorted by this point
// Adapted from: Programming Massively Parallel Processors A Hands-on Approach

__device__ void merge_sequential(int* A, int m, int* B, int n, int* C) {
    int i = 0;
    int j = 0;
    int k = 0;

    // merge while we have A and B elements
    while ((i < m) && (j < n)) { 
        if (A[i] <= B[j]) {
            C[k] = A[i];
            i++;
        } else {
            C[k] = B[j];
            j++;
        }
        k++;
    }
    // remaining elements in A
    if (i == m) {
        while (j < n) {
            C[k] = B[j];
            k++;
            j++;
        }
    } 
    // remaining elements in B
    else {
        while (i < m) {
            C[k] = A[i];
            k++;
            i++;
        }
    }
}

// main cuda kernel to merge subarray A (size m) with subarray B (size n)
// Adapted from the textbook: Programming Massively Parallel Processors A Hands-on Approach
__global__ void merge_basic_kernel(int* A, int m, int* B, int n, int* C) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int elementsPerThread = ceil((float)(m + n) / (blockDim.x * gridDim.x));

    int k_curr = tid * elementsPerThread;
    int k_next = (tid + 1) * elementsPerThread;
    
    if (m + n > k_next){
        k_next = (tid + 1) * elementsPerThread;
    }

    int i_curr = co_rank(k_curr, A, m, B, n);
    int i_next = co_rank(k_next, A, m, B, n);
    
    int j_curr = k_curr - i_curr;
    int j_next = k_next - i_next;

    merge_sequential(&A[i_curr], i_next - i_curr, &B[j_curr], j_next - j_curr, &C[k_curr]);
}

// Host function to initialize the array with random numbers
void initialize_array(int32_t* array, int size) {
    for (int i = 0; i < size; i++) {
        array[i] = rand() % 10000;
    }
}

// Helper function to print an array for sanity check
void print_array(int32_t* array, int size) {
    for (int i = 0; i < size; i++) {
        printf("%d ", array[i]);
    }
    printf("\n");
}

bool is_sorted_chunk(int32_t* chunk, int size) {
    for (int i = 1; i < size; i++) {
        if (chunk[i - 1] > chunk[i]) {
            return false;
        }
    }
    return true;
}

// checker function that checks if individual sections are sorted within the original array after the block bitonic sort, for sanity check
void check_sorted_chunks(int32_t* array, int n, int chunk_size) {
    int num_chunks = n / chunk_size;
    bool all_sorted = true;

    for (int i = 0; i < num_chunks; i++) {
        int32_t* chunk = array + i * chunk_size;  // Pointer to the start of the current chunk
        if (!is_sorted_chunk(chunk, chunk_size)) {
            //printf("Chunk %d is NOT sorted\n", i);
            all_sorted = false;
        } else {
            //printf("Chunk %d is sorted\n", i);
        }
    }

    if (all_sorted) {
        printf("All chunks are individually sorted.\n");
    } else {
        printf("Some chunks are not sorted.\n");
    }
}

// Function to check if an array is sorted in non-decreasing order, for sanity check
bool is_sorted(int32_t* array, int size) {
    for (int i = 1; i < size; i++) {
        if (array[i] < array[i - 1]) {
            //printf("%d %d %d\n", i, array[i], array[i - 1]);
            return false;
        }
    }
    return true;
}

static inline uint64_t rdtsc() {
    unsigned long a, d;
    asm volatile ("rdtsc" : "=a" (a), "=d" (d));
    return a | ((uint64_t)d<<32);
}

// Host function to perform parallel sorting and merging
void sort_array(int32_t* h_array, int n) {
    int32_t *d_array, *d_temp;
    size_t array_size = n * sizeof(int32_t);

    // Allocate memory on the device
    hipMalloc(&d_array, array_size);
    hipMalloc(&d_temp, array_size);

    // Copy input array to device
    hipMemcpy(d_array, h_array, array_size, hipMemcpyHostToDevice);

    int num_blocks = n / THREADS_PER_BLOCK;
    bitonic_block_sort<<<num_blocks, THREADS_PER_BLOCK>>>(d_array, n);
    hipDeviceSynchronize();

    int* src = d_array;
    int* dst = d_temp;
    int chunk_size = THREADS_PER_BLOCK;
    int num_chunks = n / chunk_size;

    while (num_chunks > 1) {
        int num_merges = num_chunks / 2;

        // Launch merging kernels for the current stage
        for (int i = 0; i < num_merges; i++) {
            // these are pointers to the first, second, and merged arrays that we will send to the device
            int* A_chunk = src + i * 2 * chunk_size;
            int* B_chunk = A_chunk + chunk_size;
            int* C_chunk = dst + i * 2 * chunk_size;

            // Calculate number of threads and blocks for this merge
            int num_elements = 2 * chunk_size;
            int num_threads_per_block = THREADS_PER_BLOCK;
            int num_blocks = (num_elements + num_threads_per_block - 1) / num_threads_per_block;
            //printf("CALLING KERNEL WITH %d %d %d\n", num_blocks, num_threads_per_block, num_threads_per_block * sizeof(int));
            merge_basic_kernel<<<num_blocks, num_threads_per_block>>>(A_chunk, chunk_size, B_chunk, chunk_size, C_chunk);
        }

        // Wait for all merges in this stage to complete
        hipDeviceSynchronize();

        // Swap src and dst for the next merge stage
        int* temp = src;
        src = dst;
        dst = temp;

        // Double the chunk size for the next stage
        chunk_size *= 2;
        num_chunks = num_chunks / 2;  // Round up to account for odd number of chunks
    }

    // Copy the final sorted array back to host
    hipMemcpy(h_array, src, array_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_array);
    hipFree(d_temp);
}

int main() {
    uint64_t start , end, start1, end1;
    int n = 1 << 30; 
    int32_t *sorted_array = (int32_t*)malloc(n * sizeof(int32_t));

    initialize_array(sorted_array, n);

    printf("Original Array:\n");
    print_array(sorted_array, 16); // Print first 16 elements

    // start clock
    start = rdtsc();

    start1 = clock();

    // sort array
    sort_array(sorted_array, n);

    // end clock
    end1 = clock();
    end = rdtsc();


    printf("The GPU mergesort took this many ticks: %llu \n", end - start);
    printf("The GPU mergesort took this many seconds: %f \n", (double)(end1 - start1) / CLOCKS_PER_SEC);

    printf("Sorted Array:\n");
    print_array(sorted_array, 16); // Print first 16 elements

    // Verify if the array is sorted
    if (is_sorted(sorted_array, n)) {
        //printf("The array is sorted correctly.\n");
    } else {
        printf("The array is NOT sorted correctly.\n");
    }

    free(sorted_array);
    return 0;
}
